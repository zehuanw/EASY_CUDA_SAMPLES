/********************************************************************************
 * TEX Object API 
 *
 * TODO:
 * Test the behavior of memory cache of cuArray and 2D pitched memory tex.
 * Test the behavior of float
 * I suspect some other unit can be used in analysis.
 *******************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

#define DIM 2
//#define DATA_TYPE unsigned int //please note that if change this you many also need change cudaCreateChannelDesc
//#define DATA_TYPE float //please note that if change this you many also need change cudaCreateChannelDesc
#define DATA_TYPE unsigned char //please note that if change this you many also need change cudaCreateChannelDesc
#define DIM_1 16
#define DIM_2 16
#define DIM_3 16
#define CUDA_ARRAY


#if DIM == 1
#elif DIM == 2
__global__ void kernel(hipTextureObject_t texObj){
//  DATA_TYPE a = tex2D<DATA_TYPE>(texObj,2,1);//note: require a <DATA_TYPE> in object api version.
  float a = tex2D<float>(texObj,1,0);//note: require a <DATA_TYPE> in object api version.
  //printf("%d\n",(int)a);
  printf("%f\n",a);
  return;
}
#elif DIM == 3
#endif 


int main(){
  DATA_TYPE* hostPtr = (DATA_TYPE*)malloc(DIM_1*DIM_2*DIM_3*sizeof(DATA_TYPE));
  for(int i=0;i<DIM_1*DIM_2*DIM_3;i++)
  {
    hostPtr[i] = i;
  }
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(sizeof(DATA_TYPE)*8,0,0,0,hipChannelFormatKindUnsigned);
  //cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc(sizeof(DATA_TYPE)*8,0,0,0,cudaChannelFormatKindFloat);
  
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeBorder;
  texDesc.addressMode[1] = hipAddressModeBorder;
  //texDesc.filterMode = cudaFilterModePoint;
  texDesc.filterMode = hipFilterModeLinear; //only support when cudaCreateChannelDesc ==  cudaChannelFormatKindFloat
  //texDesc.readMode = cudaReadModeElementType;
  texDesc.readMode = hipReadModeNormalizedFloat;
  texDesc.normalizedCoords = 0;
  //texDesc.normalizedCoords = 1;
  hipTextureObject_t texObj = 0;
  
  struct hipResourceDesc resDesc; //refer to the def of cudaCreateTextureObject in cuda_runtime_api.h
  memset(&resDesc, 0, sizeof(resDesc));  

  
#ifndef CUDA_ARRAY
  DATA_TYPE* devPtr;
#if DIM == 1
#elif DIM == 2
  size_t pitch;
  cudaMallocPitch(&devPtr,&pitch,DIM_1*sizeof(DATA_TYPE),DIM_2);
  cudaMemcpy2D(devPtr,pitch,hostPtr,DIM_1*sizeof(DATA_TYPE),DIM_1*sizeof(DATA_TYPE),DIM_2,cudaMemcpyHostToDevice);

  resDesc.resType = cudaResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = devPtr;
  resDesc.res.pitch2D.desc = channelDesc;
  resDesc.res.pitch2D.width = DIM_1; //should be in element not in byte
  resDesc.res.pitch2D.height = DIM_2;
  resDesc.res.pitch2D.pitchInBytes = pitch;
  cudaCreateTextureObject(&texObj,&resDesc,&texDesc,NULL);
#elif DIM == 3
#endif//#if DIM == 1
  
#else
#if DIM == 1
#elif DIM == 2
  hipArray* cuArray_2d;
  hipExtent extent_2d = {DIM_1,DIM_2,0};
  hipMalloc3DArray(&cuArray_2d, &channelDesc,extent_2d ,hipArrayDefault);//this function is able to alloc 1/3D array there are some interesting choice for the 4th parameter. Note zero in z of extent_2d.
  hipMemcpy3DParms cpy3DParms = {0}; //should be init to zero before use

  cpy3DParms.srcPtr = make_hipPitchedPtr(hostPtr,DIM_1*sizeof(DATA_TYPE),DIM_1,DIM_2);
  cpy3DParms.dstArray = cuArray_2d;
  cpy3DParms.extent = make_hipExtent(DIM_1,DIM_2,1); //If no CUDA array is participating in the copy then the extents are defined in elements of unsigned char.
  cpy3DParms.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&cpy3DParms);

  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray_2d;
  hipCreateTextureObject(&texObj,&resDesc,&texDesc,NULL);
#elif DIM == 3
#endif//#if DIM == 1
#endif//CUDA_ARRAY

  kernel<<<1,1>>>(texObj);
  hipDeviceSynchronize();
  printf("%s\n",hipGetErrorString(hipGetLastError()));

  return 0;
}
